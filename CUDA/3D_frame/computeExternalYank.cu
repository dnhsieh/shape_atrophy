#include "hip/hip_runtime.h"
#include "matvec.h"
#include "utility.h"
#include "constants.h"

__global__ void yankVolumeComputeKernel(double *d_exYElmMat, double *d_lmkEdgMat,
                                        double *d_actFcnVec, int *d_elmVtxMat, int elmNum)
{
	int elmIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( elmIdx < elmNum )
	{
		vector q10Vec, q20Vec, q30Vec;
		getEdge(q10Vec, q20Vec, q30Vec, d_lmkEdgMat, elmIdx, elmNum);

		double volVal = computeVolume(q10Vec, q20Vec, q30Vec);

		// Q = [q1 - q0, q2 - q0, q3 - q0]
		matrix QInvMat;
		matInv(QInvMat, q10Vec, q20Vec, q30Vec);

		// ---

		int q0Idx = d_elmVtxMat[             elmIdx];
		int q1Idx = d_elmVtxMat[    elmNum + elmIdx];
		int q2Idx = d_elmVtxMat[2 * elmNum + elmIdx];
		int q3Idx = d_elmVtxMat[3 * elmNum + elmIdx];

		double f0Val = d_actFcnVec[q0Idx];
		double f1Val = d_actFcnVec[q1Idx];
		double f2Val = d_actFcnVec[q2Idx];
		double f3Val = d_actFcnVec[q3Idx];
		double fcVal = (f0Val + f1Val + f2Val + f3Val) / VTXNUM;

		// ---

		vector dv0YVec, dv1YVec, dv2YVec, dv3YVec;
		dv0YVec.x = -fcVal * (-QInvMat.x.x - QInvMat.y.x - QInvMat.z.x) * volVal;
		dv0YVec.y = -fcVal * (-QInvMat.x.y - QInvMat.y.y - QInvMat.z.y) * volVal;
		dv0YVec.z = -fcVal * (-QInvMat.x.z - QInvMat.y.z - QInvMat.z.z) * volVal;
		
		dv1YVec.x = -fcVal *   QInvMat.x.x                              * volVal;
		dv1YVec.y = -fcVal *   QInvMat.x.y                              * volVal;
		dv1YVec.z = -fcVal *   QInvMat.x.z                              * volVal;

		dv2YVec.x = -fcVal *                 QInvMat.y.x                * volVal;
		dv2YVec.y = -fcVal *                 QInvMat.y.y                * volVal;
		dv2YVec.z = -fcVal *                 QInvMat.y.z                * volVal;

		dv3YVec.x = -fcVal *                               QInvMat.z.x  * volVal;
		dv3YVec.y = -fcVal *                               QInvMat.z.y  * volVal;
		dv3YVec.z = -fcVal *                               QInvMat.z.z  * volVal;

		setElement(d_exYElmMat, dv0YVec, dv1YVec, dv2YVec, dv3YVec, elmIdx, elmNum);
	}

	return;
}

//__global__ void yankBoundaryComputeKernel(double *d_exYBdrMat, double *d_lmkMat,
//                                          double *d_actFcnVec, int *d_bdrVtxMat, int lmkNum, int bdrElmNum)
//{
//	int bdrElmIdx = blockIdx.x * blockDim.x + threadIdx.x;
//	if ( bdrElmIdx < bdrElmNum )
//	{
//		int q0Idx = d_bdrVtxMat[                bdrElmIdx];
//		int q1Idx = d_bdrVtxMat[    bdrElmNum + bdrElmIdx];
//		int q2Idx = d_bdrVtxMat[2 * bdrElmNum + bdrElmIdx];
//
//		vector q0Vec, q1Vec, q2Vec;
//		getVector(q0Vec, d_lmkMat, q0Idx, lmkNum);
//		getVector(q1Vec, d_lmkMat, q1Idx, lmkNum);
//		getVector(q2Vec, d_lmkMat, q2Idx, lmkNum);
//
//		vector q10Vec, q20Vec, nmlVec;
//		vectorSubtract(q10Vec, q1Vec, q0Vec);
//		vectorSubtract(q20Vec, q2Vec, q0Vec);
//		crossProduct(nmlVec, q10Vec, q20Vec);
//
//		double f0Val = d_actFcnVec[q0Idx];
//		double f1Val = d_actFcnVec[q1Idx];
//		double f2Val = d_actFcnVec[q2Idx];
//
//		// ---	
//	
//		vector dv0YVec, dv1YVec, dv2YVec;
//
//		dv0YVec.x = f0Val * nmlVec.x / (VTXNUM - 1.0);
//		dv0YVec.y = f0Val * nmlVec.y / (VTXNUM - 1.0);
//		dv0YVec.z = f0Val * nmlVec.z / (VTXNUM - 1.0);
//
//		dv1YVec.x = f1Val * nmlVec.x / (VTXNUM - 1.0);
//		dv1YVec.y = f1Val * nmlVec.y / (VTXNUM - 1.0);
//		dv1YVec.z = f1Val * nmlVec.z / (VTXNUM - 1.0);
//
//		dv2YVec.x = f2Val * nmlVec.x / (VTXNUM - 1.0);
//		dv2YVec.y = f2Val * nmlVec.y / (VTXNUM - 1.0);
//		dv2YVec.z = f2Val * nmlVec.z / (VTXNUM - 1.0);
//
//		setBoundary(d_exYBdrMat, dv0YVec, dv1YVec, dv2YVec, bdrElmIdx, bdrElmNum);
//	}
//
//	return;
//}

__global__ void yankGatherKernel(double *d_exYMat, double *d_exYElmMat, double *d_exYBdrMat,
                                 int *d_vtxElmMat, int *d_vtxBdrMat, int elmNum, int lmkNum, int bdrElmNum)
{
	int lmkIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( lmkIdx < lmkNum )
	{
		vector exYVec = {0.0, 0.0, 0.0};

		int adjNum = d_vtxElmMat[lmkIdx];
		for ( int adjIdx = 0; adjIdx < adjNum; ++adjIdx )
		{
			int elmIdx = d_vtxElmMat[(1 + 2 * adjIdx    ) * lmkNum + lmkIdx];
			int lclIdx = d_vtxElmMat[(1 + 2 * adjIdx + 1) * lmkNum + lmkIdx];

			vector exYElmVec;
			getVector(exYElmVec, d_exYElmMat + lclIdx * elmNum * DIMNUM, elmIdx, elmNum);

			vectorSum(exYVec, exYVec, exYElmVec);
		}

		//adjNum = d_vtxBdrMat[lmkIdx];
		//for ( int adjIdx = 0; adjIdx < adjNum; ++adjIdx )
		//{
		//	int bdrElmIdx = d_vtxBdrMat[(1 + 2 * adjIdx    ) * lmkNum + lmkIdx];
		//	int    lclIdx = d_vtxBdrMat[(1 + 2 * adjIdx + 1) * lmkNum + lmkIdx];

		//	vector exYBdrVec;
		//	getVector(exYBdrVec, d_exYBdrMat + lclIdx * bdrElmNum * DIMNUM, bdrElmIdx, bdrElmNum);

		//	vectorSum(exYVec, exYVec, exYBdrVec);
		//}

		setVector(d_exYMat, exYVec, lmkIdx, lmkNum);
	}

	return;
}

void computeExternalYank(double *d_exYMat, double *d_lmkMat, double *d_lmkEdgMat, double *d_actFcnVec,
                         double *d_exYElmMat, double *d_exYBdrMat, int *d_elmVtxMat, int *d_vtxElmMat,
                         int *d_bdrVtxMat, int *d_vtxBdrMat, int lmkNum, int elmNum, int bdrElmNum)
{
	int blkNum = (elmNum - 1) / BLKDIM + 1;
	yankVolumeComputeKernel <<<blkNum, BLKDIM>>> (d_exYElmMat, d_lmkEdgMat, d_actFcnVec,
	                                              d_elmVtxMat, elmNum);

	//blkNum = (bdrElmNum - 1) / BLKDIM + 1;
	//yankBoundaryComputeKernel <<<blkNum, BLKDIM>>> (d_exYBdrMat, d_lmkMat, d_actFcnVec,
	//                                                d_bdrVtxMat, lmkNum, bdrElmNum);
	
	blkNum = (lmkNum - 1) / BLKDIM + 1;
	yankGatherKernel <<<blkNum, BLKDIM>>> (d_exYMat, d_exYElmMat, d_exYBdrMat,
	                                       d_vtxElmMat, d_vtxBdrMat, elmNum, lmkNum, bdrElmNum);

	return;
}
