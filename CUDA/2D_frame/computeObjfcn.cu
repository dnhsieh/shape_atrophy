#include <cstdio>
#include <cstdlib>
#include <cfloat>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipblas.h>
#include <hipsolver.h>
#include "struct.h"
#include "constants.h"

void assignObjfcnStructMemory(long long &, fcndata &, double *);
void objfcn(double *, double *, fcndata &);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	mxInitGPU();

	fcndata fcnObj = {0};

	mxGPUArray const *lmkIniMat;
	mxGPUArray const *elmVtxMat, *vtxElmMat, *tanVtxMat, *tsvVtxMat, *femVtxMat, *femIfoMat;
	mxGPUArray const *bdrVtxMat, *vtxBdrMat, *btmVtxMat, *vtxBtmMat, *vfdVtxMat;
	mxGPUArray const *tgtCenPosMat, *tgtUniDirMat, *tgtElmVolVec;

	double *h_tauVarMat       = (double *) mxGetDoubles(prhs[ 0]);
	fcnObj.prm.h_tauPrmVec    = (double *) mxGetDoubles(prhs[ 1]);
	lmkIniMat                 =  mxGPUCreateFromMxArray(prhs[ 2]);
	elmVtxMat                 =  mxGPUCreateFromMxArray(prhs[ 3]);
	vtxElmMat                 =  mxGPUCreateFromMxArray(prhs[ 4]);
	tanVtxMat                 =  mxGPUCreateFromMxArray(prhs[ 5]);
	tsvVtxMat                 =  mxGPUCreateFromMxArray(prhs[ 6]);
	bdrVtxMat                 =  mxGPUCreateFromMxArray(prhs[ 7]);
	vtxBdrMat                 =  mxGPUCreateFromMxArray(prhs[ 8]);
	btmVtxMat                 =  mxGPUCreateFromMxArray(prhs[ 9]);
	vtxBtmMat                 =  mxGPUCreateFromMxArray(prhs[10]);
	femVtxMat                 =  mxGPUCreateFromMxArray(prhs[11]);
	femIfoMat                 =  mxGPUCreateFromMxArray(prhs[12]);
	vfdVtxMat                 =  mxGPUCreateFromMxArray(prhs[13]);
	tgtCenPosMat              =  mxGPUCreateFromMxArray(prhs[14]);
	tgtUniDirMat              =  mxGPUCreateFromMxArray(prhs[15]);
	tgtElmVolVec              =  mxGPUCreateFromMxArray(prhs[16]);
	fcnObj.vfd.cenKnlType     =             mxGetScalar(prhs[17]);
	fcnObj.vfd.cenKnlWidth    =             mxGetScalar(prhs[18]);
	fcnObj.vfd.dirKnlType     =             mxGetScalar(prhs[19]);
	fcnObj.vfd.dirKnlWidth    =             mxGetScalar(prhs[20]);
	fcnObj.prm.ldmWgt         =             mxGetScalar(prhs[21]);
	fcnObj.prm.knlOrder       =             mxGetScalar(prhs[22]);
	fcnObj.prm.knlWidth       =             mxGetScalar(prhs[23]);
	fcnObj.prm.knlEps         =             mxGetScalar(prhs[24]);
	fcnObj.prm.h_modVec       = (double *) mxGetDoubles(prhs[25]);
	fcnObj.prm.spdTanVal      =             mxGetScalar(prhs[26]);
	fcnObj.prm.spdTsvVal      =             mxGetScalar(prhs[27]);
	fcnObj.prm.h_ynkActPrmVec = (double *) mxGetDoubles(prhs[28]);
	fcnObj.prm.h_reaActPrmVec = (double *) mxGetDoubles(prhs[29]);
	fcnObj.prm.btmWgt         =             mxGetScalar(prhs[30]);
	fcnObj.prm.timeStp        =             mxGetScalar(prhs[31]);
	fcnObj.prm.timeNum        =             mxGetScalar(prhs[32]);
	fcnObj.pcg.itrMax         =             mxGetScalar(prhs[33]);
	fcnObj.pcg.tolSqu         =             mxGetScalar(prhs[34]);

	int objVarNum = mxGetM(prhs[0]);
	int objTotNum = mxGetN(prhs[0]);

	plhs[0] = mxCreateDoubleMatrix(1, objTotNum, mxREAL);
	double *h_objVec = (double *) mxGetDoubles(plhs[0]);

	// ---

	fcnObj.prm.d_lmkIniMat = (double *) mxGPUGetDataReadOnly(lmkIniMat);
	fcnObj.elm.d_elmVtxMat = (int    *) mxGPUGetDataReadOnly(elmVtxMat);
	fcnObj.elm.d_vtxElmMat = (int    *) mxGPUGetDataReadOnly(vtxElmMat);
	fcnObj.elm.d_tanVtxMat = (int    *) mxGPUGetDataReadOnly(tanVtxMat);
	fcnObj.elm.d_tsvVtxMat = (int    *) mxGPUGetDataReadOnly(tsvVtxMat);
	fcnObj.elm.d_bdrVtxMat = (int    *) mxGPUGetDataReadOnly(bdrVtxMat);
	fcnObj.elm.d_vtxBdrMat = (int    *) mxGPUGetDataReadOnly(vtxBdrMat);
	fcnObj.elm.d_btmVtxMat = (int    *) mxGPUGetDataReadOnly(btmVtxMat);
	fcnObj.elm.d_vtxBtmMat = (int    *) mxGPUGetDataReadOnly(vtxBtmMat);
	fcnObj.elm.d_femVtxMat = (int    *) mxGPUGetDataReadOnly(femVtxMat);
	fcnObj.elm.d_femIfoMat = (int    *) mxGPUGetDataReadOnly(femIfoMat);
	fcnObj.elm.d_vfdVtxMat = (int    *) mxGPUGetDataReadOnly(vfdVtxMat);
	fcnObj.tgt.d_cenPosMat = (double *) mxGPUGetDataReadOnly(tgtCenPosMat);
	fcnObj.tgt.d_uniDirMat = (double *) mxGPUGetDataReadOnly(tgtUniDirMat);
	fcnObj.tgt.d_elmVolVec = (double *) mxGPUGetDataReadOnly(tgtElmVolVec);

	mwSize const *lmkDims    = mxGPUGetDimensions(lmkIniMat);
	mwSize const *elmDims    = mxGPUGetDimensions(elmVtxMat);
	mwSize const *nzrDims    = mxGPUGetDimensions(femVtxMat);
	mwSize const *bdrElmDims = mxGPUGetDimensions(bdrVtxMat);
	mwSize const *btmLmkDims = mxGPUGetDimensions(vtxBtmMat);
	mwSize const *btmElmDims = mxGPUGetDimensions(btmVtxMat);
	mwSize const *vfdElmDims = mxGPUGetDimensions(vfdVtxMat);
	mwSize const *tgtElmDims = mxGPUGetDimensions(tgtCenPosMat);

	fcnObj.prm.varNum    = objVarNum;
	fcnObj.prm.lmkNum    =    lmkDims[0];
	fcnObj.prm.elmNum    =    elmDims[0];
	fcnObj.prm.nzrNum    =    nzrDims[0];
	fcnObj.prm.bdrElmNum = bdrElmDims[0];
	fcnObj.prm.btmLmkNum = btmLmkDims[0];
	fcnObj.prm.btmElmNum = btmElmDims[0];
	fcnObj.prm.vfdLmkNum = fcnObj.prm.lmkNum;
	fcnObj.prm.vfdElmNum = vfdElmDims[0];
	fcnObj.tgt.vfdElmNum = tgtElmDims[0];

	fcnObj.pcg.varNum = fcnObj.prm.lmkNum * DIMNUM;

	// ---

	int lmkNum    = fcnObj.prm.lmkNum;
	int elmNum    = fcnObj.prm.elmNum;
	int bdrElmNum = fcnObj.prm.bdrElmNum;
	int btmElmNum = fcnObj.prm.btmElmNum;
	int vfdElmNum = fcnObj.prm.vfdElmNum;
	int timeNum   = fcnObj.prm.timeNum;

	long long gpuAloDblMemCnt =  fcnObj.pcg.varNum * 6
	                           + lmkNum    * (lmkNum * 8 + DIMNUM * 3 + DIMNUM * timeNum * 2 + timeNum + 3)
	                           + elmNum    * (DIMNUM * 2 + DIMNUM * VTXNUM * 5 + DIMNUM * DIMNUM
	                                          + (1 + VTXNUM) * VTXNUM / 2 + 1)
	                           + bdrElmNum * DIMNUM * (VTXNUM - 1)
	                           + btmElmNum * 2 * DIMNUM * (VTXNUM - 1)
	                           + vfdElmNum * (DIMNUM * 2 + 2) + fcnObj.tgt.vfdElmNum 
	                           + SUMBLKDIM;

	double *gpuDblSpace;
	hipError_t error = hipMalloc((void **) &gpuDblSpace, sizeof(double) * gpuAloDblMemCnt);
	if ( error != hipSuccess )
		mexErrMsgIdAndTxt("objfcn2Dframe:hipMalloc", "Fail to allocate device memory.");

	hipMalloc((void **) &(fcnObj.d_status), sizeof(int));

	long long gpuAsgDblMemCnt;
	assignObjfcnStructMemory(gpuAsgDblMemCnt, fcnObj, gpuDblSpace);
	if ( gpuAsgDblMemCnt != gpuAloDblMemCnt )
	{
		mexErrMsgIdAndTxt("objfcn2Dframe:memAssign", 
		                  "Assigned device double memory (%lld) mismatches the allocated memory (%lld).", 
		                  gpuAsgDblMemCnt, gpuAloDblMemCnt);
	}

	// ---

	hipblasCreate(&(fcnObj.blasHdl));

	hipsolverDnCreate(&(fcnObj.solvHdl));
	hipsolverDnDpotrf_bufferSize(fcnObj.solvHdl, HIPBLAS_FILL_MODE_LOWER, fcnObj.prm.lmkNum, fcnObj.d_knlMat,
	                            fcnObj.prm.lmkNum, &(fcnObj.h_Lwork));

	hipMalloc((void **) &(fcnObj.d_workspace), sizeof(double) * fcnObj.h_Lwork);

	// ---

	hipEvent_t tic, toc;
	float       timeRun;

	hipEventCreate(&tic);
	hipEventCreate(&toc);

	// ---

	hipEventRecord(tic);
	for ( int objIdx = 0; objIdx < objTotNum; ++objIdx )
	{
		double *h_posPtr = h_tauVarMat + objIdx * objVarNum;
		objfcn(h_objVec + objIdx, h_posPtr, fcnObj);
	}
	hipEventRecord(toc);
	hipEventSynchronize(toc);
	hipEventElapsedTime(&timeRun, tic, toc);

	printf("Time %f msec\n", timeRun);

	// ---
	//

	mxGPUDestroyGPUArray(lmkIniMat);
	mxGPUDestroyGPUArray(elmVtxMat);
	mxGPUDestroyGPUArray(vtxElmMat);
	mxGPUDestroyGPUArray(tanVtxMat);
	mxGPUDestroyGPUArray(tsvVtxMat);
	mxGPUDestroyGPUArray(bdrVtxMat);
	mxGPUDestroyGPUArray(vtxBdrMat);
	mxGPUDestroyGPUArray(btmVtxMat);
	mxGPUDestroyGPUArray(vtxBtmMat);
	mxGPUDestroyGPUArray(femVtxMat);
	mxGPUDestroyGPUArray(femIfoMat);
	mxGPUDestroyGPUArray(vfdVtxMat);
	mxGPUDestroyGPUArray(tgtCenPosMat);
	mxGPUDestroyGPUArray(tgtUniDirMat);
	mxGPUDestroyGPUArray(tgtElmVolVec);

	mxFree((void *) lmkDims);
	mxFree((void *) elmDims);
	mxFree((void *) nzrDims);
	mxFree((void *) bdrElmDims);
	mxFree((void *) btmLmkDims);
	mxFree((void *) btmElmDims);
	mxFree((void *) vfdElmDims);
	mxFree((void *) tgtElmDims);

	hipFree(gpuDblSpace);
	hipFree(fcnObj.d_status);
	hipFree(fcnObj.d_workspace);

	hipblasDestroy(fcnObj.blasHdl);
	hipsolverDnDestroy(fcnObj.solvHdl);

	hipEventDestroy(tic);
	hipEventDestroy(toc);

	return;
}

