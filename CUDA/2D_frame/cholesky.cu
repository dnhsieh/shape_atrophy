#include <hipsolver.h>
#include "struct.h"

void cholesky(double *d_knlMat, fcndata &fcnObj) 
{
	int lmkNum = fcnObj.prm.lmkNum;

	hipsolverHandle_t solvHdl     = fcnObj.solvHdl;
	int                h_Lwork     = fcnObj.h_Lwork;
	double            *d_workspace = fcnObj.d_workspace;
	int               *d_status    = fcnObj.d_status;

	hipsolverDnDpotrf(solvHdl, HIPBLAS_FILL_MODE_LOWER, 
	                 lmkNum, d_knlMat, lmkNum, d_workspace, h_Lwork, d_status);

	return;
}
