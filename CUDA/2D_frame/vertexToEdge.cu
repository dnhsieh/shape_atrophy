#include "hip/hip_runtime.h"
#include "matvec.h"
#include "constants.h"

__global__ void vertexToEdgeKernel(double *d_edgMat, double *d_datMat,
                                   int *d_elmVtxMat, int datNum, int elmNum)
{
	int elmIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( elmIdx < elmNum )
	{
		int q0Idx = d_elmVtxMat[             elmIdx];
		int q1Idx = d_elmVtxMat[    elmNum + elmIdx];
		int q2Idx = d_elmVtxMat[2 * elmNum + elmIdx];

		vector q0Vec, q1Vec, q2Vec;
		getVector(q0Vec, d_datMat, q0Idx, datNum);
		getVector(q1Vec, d_datMat, q1Idx, datNum);
		getVector(q2Vec, d_datMat, q2Idx, datNum);

		vector q10Vec, q20Vec;
		vectorSubtract(q10Vec, q1Vec, q0Vec);
		vectorSubtract(q20Vec, q2Vec, q0Vec);
		
		setEdge(d_edgMat, q10Vec, q20Vec, elmIdx, elmNum);
	}

	return;
}

void vertexToEdge(double *d_edgMat, double *d_datMat, int *d_elmVtxMat, int datNum, int elmNum)
{
	int blkNum = (elmNum - 1) / BLKDIM + 1;
	vertexToEdgeKernel <<<blkNum, BLKDIM>>> (d_edgMat, d_datMat, d_elmVtxMat, datNum, elmNum);

	return;
}
