#include "hip/hip_runtime.h"
#include "matvec.h"
#include "constants.h"

__global__ void penaltyComputeKernel(double *d_pnlBtmMat, double *d_lmkBtmMat, double *d_vlcMat,
                                     int *d_btmVtxMat, int lmkNum, int btmElmNum)
{
	int btmElmIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( btmElmIdx < btmElmNum )
	{
		vector q0Vec, q1Vec;
		getBoundary(q0Vec, q1Vec, d_lmkBtmMat, btmElmIdx, btmElmNum);

		int q0Idx = d_btmVtxMat[            btmElmIdx];
		int q1Idx = d_btmVtxMat[btmElmNum + btmElmIdx];

		vector v0Vec, v1Vec;
		getVector(v0Vec, d_vlcMat, q0Idx, lmkNum);
		getVector(v1Vec, d_vlcMat, q1Idx, lmkNum);

		vector tanVec;
		vectorSubtract(tanVec, q1Vec, q0Vec);

		double tanLen = eucnorm(tanVec);

		vector dv0PVec, dv1PVec;

		dv0PVec.x = v0Vec.x * tanLen / (VTXNUM - 1.0);
		dv0PVec.y = v0Vec.y * tanLen / (VTXNUM - 1.0);

		dv1PVec.x = v1Vec.x * tanLen / (VTXNUM - 1.0);
		dv1PVec.y = v1Vec.y * tanLen / (VTXNUM - 1.0);

		setBoundary(d_pnlBtmMat, dv0PVec, dv1PVec, btmElmIdx, btmElmNum);
	}

	return;
}

__global__ void penaltyGatherKernel(double *d_pnlMat, double *d_pnlBtmMat, int *d_vtxBtmMat, 
                                    int btmElmNum, int lmkNum, int btmLmkNum)
{
	int btmLmkIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( btmLmkIdx < btmLmkNum )
	{
		vector pnlVec = {0.0, 0.0};

		int adjNum = d_vtxBtmMat[btmLmkIdx];
		for ( int adjIdx = 0; adjIdx < adjNum; ++adjIdx )
		{
			int btmElmIdx = d_vtxBtmMat[(1 + 2 * adjIdx    ) * btmLmkNum + btmLmkIdx];
			int    lclIdx = d_vtxBtmMat[(1 + 2 * adjIdx + 1) * btmLmkNum + btmLmkIdx];

			vector pnlBtmVec;
			getVector(pnlBtmVec, d_pnlBtmMat + lclIdx * btmElmNum * DIMNUM, btmElmIdx, btmElmNum);

			vectorSum(pnlVec, pnlVec, pnlBtmVec);
		}

		setVector(d_pnlMat, pnlVec, btmLmkIdx, lmkNum);
	}

	return;
}

void applyPenaltyOperator(double *d_pnlMat, double *d_lmkBtmMat, double *d_vlcMat,
                          double *d_pnlBtmMat, int *d_btmVtxMat, int *d_vtxBtmMat,
                          int lmkNum, int btmElmNum, int btmLmkNum) 
{
	int blkNum = (btmElmNum - 1) / BLKDIM + 1;
	penaltyComputeKernel <<<blkNum, BLKDIM>>> (d_pnlBtmMat, d_lmkBtmMat, d_vlcMat,
	                                           d_btmVtxMat, lmkNum, btmElmNum);

	blkNum = (btmLmkNum - 1) / BLKDIM + 1;
	hipMemset(d_pnlMat, 0, sizeof(double) * lmkNum * DIMNUM);
	penaltyGatherKernel <<<blkNum, BLKDIM>>> (d_pnlMat, d_pnlBtmMat,
	                                          d_vtxBtmMat, btmElmNum, lmkNum, btmLmkNum);

	return;
}
