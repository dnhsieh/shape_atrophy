#include "hip/hip_runtime.h"
#include "matvec.h"
#include "constants.h"

__global__ void vertexToBottomKernel(double *d_btmMat, double *d_datMat,
                                     int *d_btmVtxMat, int datNum, int btmElmNum)
{
	int btmElmIdx = blockIdx.x * blockDim.x + threadIdx.x;
	if ( btmElmIdx < btmElmNum )
	{
		int q0Idx = d_btmVtxMat[            btmElmIdx];
		int q1Idx = d_btmVtxMat[btmElmNum + btmElmIdx];

		vector q0Vec, q1Vec;
		getVector(q0Vec, d_datMat, q0Idx, datNum);
		getVector(q1Vec, d_datMat, q1Idx, datNum);

		setBoundary(d_btmMat, q0Vec, q1Vec, btmElmIdx, btmElmNum);
	}

	return;
}

void vertexToBottom(double *d_btmMat, double *d_datMat, int *d_btmVtxMat,
                    int datNum, int btmElmNum)
{
	int blkNum = (btmElmNum - 1) / BLKDIM + 1;
	vertexToBottomKernel <<<blkNum, BLKDIM>>> (d_btmMat, d_datMat, d_btmVtxMat, datNum, btmElmNum);

	return;
}
